#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -(10*waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
		SwitchToThread();
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#else
#include <unistd.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
	usleep(waitTime);
}
#endif

#include "cryptonight.h"
#include "cuda_extra.h"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"

#ifdef XMRMINER_LARGEGRID
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

__device__ __forceinline__ uint64_t cuda_mul128( uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi )
{
	*product_hi = __umul64hi( multiplier, multiplicand );
	return (multiplier * multiplicand );
}

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : "l"( addr ) );
	return x;
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : "l"( addr ) );
	return x;
}


template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
	asm volatile( "st.global.cg.u32 [%0], %1;" : : "l"( addr ), "r"( val ) );
}

__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	const int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = 0x80000 >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];

	MEMCPY8( key, ctx_key1 + thread * 40, 20 );

	if( partidx == 0 )
	{
		// first round
		MEMCPY8( text, ctx_state + thread * 50 + sub + 16, 2 );
	}
	else
	{
		// load previous text data
		MEMCPY8( text, &long_state[( (uint64_t) thread << 19 ) + sub + start - 32], 2 );
	}
	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
		cn_aes_pseudo_round_mut( sharedMemory, text, key );
		MEMCPY8(&long_state[((uint64_t) thread << 19) + (sub + i)], text, 2);
	}
}

#ifdef XMR_THREADS
__launch_bounds__( XMRMINER_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2( int threads, int bfactor, int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );

	__syncthreads( );

#if __CUDA_ARCH__ >= 300

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
	const int sub = threadIdx.x & 3;
	const int sub2 = sub & 2;

	if ( thread >= threads )
		return;

	int i, k, j;
	const int batchsize = ITER >> ( 2 + bfactor );
	const int start = partidx * batchsize;
	const int end = start + batchsize;
	uint32_t * long_state = &d_long_state[(IndexType) thread << 19];
	uint32_t * ctx_a = d_ctx_a + thread * 4;
	uint32_t * ctx_b = d_ctx_b + thread * 4;
	uint32_t a, d[2];
	uint32_t t1[2], t2[2], res;
	uint64_t reshi, reslo;

	a = ctx_a[sub];
	d[1] = ctx_b[sub];
	#pragma unroll 2
	for ( i = start; i < end; ++i )
	{
		#pragma unroll 2
		for ( int x = 0; x < 2; ++x )
		{

			j = ( ( ( __shfl( (int) a, 0, 4 ) & 0x1FFFF0 ) >> 2 ) + sub );

			const int x_0 = loadGlobal32<uint32_t>( long_state + j );
			const uint32_t x_1 = __shfl( x_0, sub + 1, 4 );
			const uint32_t x_2 = __shfl( x_0, sub + 2, 4 );
			const uint32_t x_3 = __shfl( x_0, sub + 3, 4 );
			d[x] = a ^
				t_fn0( x_0 & 0xff ) ^
				t_fn1( (x_1 >> 8) & 0xff ) ^
				t_fn2( (x_2 >> 16) & 0xff ) ^
				t_fn3( ( x_3 >> 24 ) );


			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			t1[0] = __shfl( (int) d[x], 0, 4 );
			//long_state[j] = d[0] ^ d[1];
			storeGlobal32( long_state + j, d[0] ^ d[1] );

			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & 0x1FFFF0]);
			j = ( ( *t1 & 0x1FFFF0 ) >> 2 ) + sub;

			uint32_t yy[2];
			*( (uint64_t*) yy ) = loadGlobal64<uint64_t>( ( (uint64_t *) long_state )+( j >> 1 ) );
			uint32_t zz[2];
			zz[0] = __shfl( yy[0], 0, 4 );
			zz[1] = __shfl( yy[1], 0, 4 );

			t1[1] = __shfl( (int) d[x], 1, 4 );
			#pragma unroll
			for ( k = 0; k < 2; k++ )
				t2[k] = __shfl( (int) a, k + sub2, 4 );
			asm(
				"mad.lo.u64 %0, %2, %3, %4;\n\t"
				"mad.hi.u64 %1, %2, %3, %4;\n\t"
				 : "=l"( reslo ), "=l"( reshi )
				: "l"( *( (uint64_t *) t1 ) ), "l"( *( (uint64_t*) zz ) ), "l"( *( (uint64_t *) t2 ) ) );
			res = ( sub2 ? reslo : reshi ) >> ( sub & 1 ? 32 : 0 );

			storeGlobal32( long_state + j, res );
			a = ( sub & 1 ? yy[1] : yy[0] ) ^ res;
		}
	}

	if ( bfactor > 0 )
	{
		ctx_a[sub] = a;
		ctx_b[sub] = d[1];
	}

#else // __CUDA_ARCH__ < 300

	const int thread = blockDim.x * blockIdx.x + threadIdx.x;

	if ( thread >= threads )
		return;

	int i, j;
	const int batchsize = ITER >> ( 2 + bfactor );
	const int start = partidx * batchsize;
	const int end = start + batchsize;
	uint32_t * __restrict__ long_state = &d_long_state[(IndexType) thread << 19];
	uint32_t * __restrict__ ctx_a = d_ctx_a + thread * 4;
	uint32_t * __restrict__ ctx_b = d_ctx_b + thread * 4;
	uint32_t a[4], b[4], c[4];

	MEMCPY8( a, ctx_a, 2 );
	MEMCPY8( b, ctx_b, 2 );

	for ( i = start; i < end; ++i )
	{
		j = ( a[0] & 0x1FFFF0 ) >> 2;
		cn_aes_single_round( sharedMemory, &long_state[j], c, a );
		XOR_BLOCKS_DST( c, b, &long_state[j] );
		MUL_SUM_XOR_DST( c, a, &long_state[( c[0] & 0x1FFFF0 ) >> 2] );
		j = ( a[0] & 0x1FFFF0 ) >> 2;
		cn_aes_single_round( sharedMemory, &long_state[j], b, a );
		XOR_BLOCKS_DST( b, c, &long_state[j] );
		MUL_SUM_XOR_DST( b, a, &long_state[( b[0] & 0x1FFFF0 ) >> 2] );
	}

	if ( bfactor > 0 )
	{
		MEMCPY8( ctx_a, a, 2 );
		MEMCPY8( ctx_b, b, 2 );
	}

#endif // __CUDA_ARCH__ >= 300
}

__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = 0x80000 >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];
	MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
	MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
#pragma unroll
		for ( int j = 0; j < 4; ++j )
			text[j] ^= long_state[((IndexType) thread << 19) + (sub + i + j)];

		cn_aes_pseudo_round_mut( sharedMemory, text, key );
	}

	MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}

extern "C" void cryptonight_core_cpu_hash(nvid_ctx* ctx)
{
	dim3 grid( ctx->device_blocks );
	dim3 block( ctx->device_threads );
	dim3 block4( ctx->device_threads << 2 );
	dim3 block8( ctx->device_threads << 3 );

	int partcount = 1 << ctx->device_bfactor;

	/* bfactor for phase 1 and 3
	 *
	 * phase 1 and 3 consume less time than phase 2, therefore we begin with the
	 * kernel splitting if the user defined a `bfactor >= 5`
	 */
	int bfactorOneThree = ctx->device_bfactor - 4;
	if( bfactorOneThree < 0 )
		bfactorOneThree = 0;

	int partcountOneThree = 1 << bfactorOneThree;

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		cryptonight_core_gpu_phase1<<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state, ctx->d_ctx_state, ctx->d_ctx_key1 );
		exit_if_cudaerror( ctx->device_id, __FILE__, __LINE__ );

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}
	if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );

	for ( int i = 0; i < partcount; i++ )
	{
		cryptonight_core_gpu_phase2<<< grid, ( ctx->device_arch[0] >= 3 ? block4 : block ) >>>( ctx->device_blocks*ctx->device_threads,
			ctx->device_bfactor, i, ctx->d_long_state, ctx->d_ctx_a, ctx->d_ctx_b );
		exit_if_cudaerror( ctx->device_id, __FILE__, __LINE__ );

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		cryptonight_core_gpu_phase3<<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state,
			ctx->d_ctx_state, ctx->d_ctx_key2 );
		exit_if_cudaerror( ctx->device_id, __FILE__, __LINE__ );
	}
}
